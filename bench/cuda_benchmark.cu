#include "cuda_benchmark.h"

#include <hipcub/hipcub.hpp>

CudaBenchmark::CudaBenchmark() {
  hipStreamCreate(&stream_);
  hipEventCreate(&start_timestamp_);
  hipEventCreate(&end_timestamp_);

  // allocate in/out memories
  hipMalloc(&keys_ptr_, MAX_ELEMENT_COUNT * sizeof(uint32_t));
  hipMalloc(&values_ptr_, MAX_ELEMENT_COUNT * sizeof(uint32_t));
  hipMalloc(&out_keys_ptr_, MAX_ELEMENT_COUNT * sizeof(uint32_t));
  hipMalloc(&out_values_ptr_, MAX_ELEMENT_COUNT * sizeof(uint32_t));
}

CudaBenchmark::~CudaBenchmark() {
  hipStreamDestroy(stream_);
  hipEventDestroy(start_timestamp_);
  hipEventDestroy(end_timestamp_);

  if (temp_storage_) hipFree(temp_storage_);
  hipFree(keys_ptr_);
  hipFree(values_ptr_);
  hipFree(out_keys_ptr_);
  hipFree(out_values_ptr_);
}

CudaBenchmark::Results CudaBenchmark::Sort(const std::vector<uint32_t> &keys) {
  auto n = keys.size();

  Results result;
  result.keys.resize(n);

  // CPU to GPU
  hipMemcpy(keys_ptr_, keys.data(), n * sizeof(uint32_t), hipMemcpyDefault);

  // allocate temp storage
  size_t temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortKeys(nullptr, temp_storage_bytes,
                                 static_cast<const uint32_t *>(keys_ptr_),
                                 static_cast<uint32_t *>(out_keys_ptr_), n);
  if (temp_storage_bytes_ < temp_storage_bytes) {
    if (temp_storage_) hipFree(temp_storage_);
    hipMalloc(&temp_storage_, temp_storage_bytes);
    temp_storage_bytes_ = temp_storage_bytes;
  }

  // cub sort, measure time
  hipStreamSynchronize(stream_);
  hipEventRecord(start_timestamp_, stream_);
  hipcub::DeviceRadixSort::SortKeys(temp_storage_, temp_storage_bytes_,
                                 static_cast<const uint32_t *>(keys_ptr_),
                                 static_cast<uint32_t *>(out_keys_ptr_), n, 0,
                                 32, stream_);
  hipEventRecord(end_timestamp_, stream_);
  hipStreamSynchronize(stream_);

  // GPU to CPU
  hipMemcpy(result.keys.data(), out_keys_ptr_, n * sizeof(uint32_t),
             hipMemcpyDefault);

  // measure time
  float ms;
  hipEventElapsedTime(&ms, start_timestamp_, end_timestamp_);
  result.total_time = static_cast<uint64_t>(ms * 1e6);  // ms to ns

  return result;
}

CudaBenchmark::Results CudaBenchmark::SortKeyValue(
    const std::vector<uint32_t> &keys, const std::vector<uint32_t> &values) {
  auto n = keys.size();

  Results result;
  result.keys.resize(n);
  result.values.resize(n);

  // CPU to GPU
  hipMemcpy(keys_ptr_, keys.data(), n * sizeof(uint32_t), hipMemcpyDefault);
  hipMemcpy(values_ptr_, values.data(), n * sizeof(uint32_t),
             hipMemcpyDefault);

  // allocate temp storage
  size_t temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairs(nullptr, temp_storage_bytes,
                                  static_cast<const uint32_t *>(keys_ptr_),
                                  static_cast<uint32_t *>(out_keys_ptr_),
                                  static_cast<const uint32_t *>(values_ptr_),
                                  static_cast<uint32_t *>(out_values_ptr_), n);
  if (temp_storage_bytes_ < temp_storage_bytes) {
    if (temp_storage_) hipFree(temp_storage_);
    hipMalloc(&temp_storage_, temp_storage_bytes);
    temp_storage_bytes_ = temp_storage_bytes;
  }

  // cub sort, measure time
  hipStreamSynchronize(stream_);
  hipEventRecord(start_timestamp_, stream_);
  hipcub::DeviceRadixSort::SortPairs(temp_storage_, temp_storage_bytes,
                                  static_cast<const uint32_t *>(keys_ptr_),
                                  static_cast<uint32_t *>(out_keys_ptr_),
                                  static_cast<const uint32_t *>(values_ptr_),
                                  static_cast<uint32_t *>(out_values_ptr_), n,
                                  0, 32, stream_);
  hipEventRecord(end_timestamp_, stream_);
  hipStreamSynchronize(stream_);

  // GPU to CPU
  hipMemcpy(result.keys.data(), out_keys_ptr_, n * sizeof(uint32_t),
             hipMemcpyDefault);
  hipMemcpy(result.values.data(), out_values_ptr_, n * sizeof(uint32_t),
             hipMemcpyDefault);

  // measure time
  float ms;
  hipEventElapsedTime(&ms, start_timestamp_, end_timestamp_);
  result.total_time = static_cast<uint64_t>(ms * 1e6);  // ms to ns
  return result;
}
